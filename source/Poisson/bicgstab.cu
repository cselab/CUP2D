#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipsparse.h"

#include "hip/hip_runtime_api.h"
#include "bicgstab.h"

extern "C" void BiCGSTAB(
    const int m, // rows
    const int n, // cols
    const int nnz, // no. of non-zero elements
    double* const h_cooValA,
    int* const h_cooRowA,
    int* const h_cooColA,
    double* const h_x, // contains initial guess
    double* const h_b,
    const double max_error,
    const double max_rel_error,
    const int max_restarts) // Defaults to normal BiCGSTAB without tricks
{
  // --------------------------------------------- Set-up streams and handles ---------------------------------------
  hipStream_t solver_stream;
  hipblasHandle_t cublas_handle;
  hipsparseHandle_t cusparse_handle;
  checkCudaErrors(hipStreamCreate(&solver_stream));
  checkCudaErrors(hipblasCreate(&cublas_handle)); 
  checkCudaErrors(hipsparseCreate(&cusparse_handle)); 
  // Set handles to stream
  checkCudaErrors(hipblasSetStream(cublas_handle, solver_stream));
  checkCudaErrors(hipsparseSetStream(cusparse_handle, solver_stream));

  // ------------------------------------------------- H2D transfer --------------------------------------------------
  // Host-device exec asynchronous, it may be worth already allocating pinned memory
  // and copying h2h (with cpu code) after async dev memory allocation calls 
  // to speed up h2d transfer down the line
   
  // Allocate device memory for linear system
  double* d_cooValA = NULL;
  double* d_cooValA_sorted = NULL;
  int* d_cooRowA = NULL;
  int* d_cooColA = NULL;
  double* d_x = NULL;
  double* d_b = NULL;
  checkCudaErrors(hipMallocAsync(&d_cooValA, nnz * sizeof(double), solver_stream));
  checkCudaErrors(hipMallocAsync(&d_cooValA_sorted, nnz * sizeof(double), solver_stream));
  checkCudaErrors(hipMallocAsync(&d_cooRowA, nnz * sizeof(int), solver_stream));
  checkCudaErrors(hipMallocAsync(&d_cooColA, nnz * sizeof(int), solver_stream));
  checkCudaErrors(hipMallocAsync(&d_x, m * sizeof(double), solver_stream));
  checkCudaErrors(hipMallocAsync(&d_b, m * sizeof(double), solver_stream));

  // Possibly copy to pinned memory here followed by a sync call

  // H2D transfer of linear system
  checkCudaErrors(hipMemcpyAsync(d_cooValA, h_cooValA, nnz * sizeof(double), hipMemcpyHostToDevice, solver_stream));
  checkCudaErrors(hipMemcpyAsync(d_cooRowA, h_cooRowA, nnz * sizeof(int), hipMemcpyHostToDevice, solver_stream));
  checkCudaErrors(hipMemcpyAsync(d_cooColA, h_cooColA, nnz * sizeof(int), hipMemcpyHostToDevice, solver_stream));
  checkCudaErrors(hipMemcpyAsync(d_x, h_x, m * sizeof(double), hipMemcpyHostToDevice, solver_stream));
  checkCudaErrors(hipMemcpyAsync(d_b, h_b, m * sizeof(double), hipMemcpyHostToDevice, solver_stream));
  
  // Sort COO storage by row
  // 1. Deduce buffer size necessary for sorting and allocate storage for it
  size_t coosortBuffSz;
  void* coosortBuff;
  checkCudaErrors(hipsparseXcoosort_bufferSizeExt(cusparse_handle, m, n, nnz, d_cooRowA, d_cooColA, &coosortBuffSz));
  checkCudaErrors(hipMallocAsync(&coosortBuff, coosortBuffSz * sizeof(char), solver_stream));

  // 2. Set-up permutation vector P to track transformation from un-sorted to sorted list
  int* d_P;
  checkCudaErrors(hipMallocAsync(&d_P, nnz * sizeof(int), solver_stream));
  checkCudaErrors(hipsparseCreateIdentityPermutation(cusparse_handle, nnz, d_P));

  // 3. Sort d_cooRowA_ and d_cooCol inplace and apply permutation stored in d_P to d_cooValA_
  checkCudaErrors(hipsparseXcoosortByRow(cusparse_handle, m, n, nnz, d_cooRowA, d_cooColA, d_P, coosortBuff));
  checkCudaErrors(hipsparseDgthr(cusparse_handle, nnz, d_cooValA, d_cooValA_sorted, d_P, HIPSPARSE_INDEX_BASE_ZERO));

  // Free buffers allocated for COO sort
  checkCudaErrors(hipFreeAsync(coosortBuff, solver_stream));
  checkCudaErrors(hipFreeAsync(d_P, solver_stream));

  // ---------------------------------------------- BiCGSTAB ----------------------------------------------------------
  const double eye = 1.;
  const double nye = -1.;
  const double nil = 0.;

  /*
    This function generally follows notation of the Wikipedia page with several omissions
    to increase variable reuse.  Specifically:
      - d_x <-> h, x_i
      - d_b <-> r_0, r_i, s
  */

  // Initialize BiCGSTAB arrays and allocate memory
  double* d_rhat = NULL;
  double* d_p = NULL;
  double* d_nu = NULL;
  double* d_t = NULL;
  checkCudaErrors(hipMallocAsync(&d_rhat, m * sizeof(double), solver_stream));
  checkCudaErrors(hipMallocAsync(&d_p, m * sizeof(double), solver_stream));
  checkCudaErrors(hipMallocAsync(&d_nu, m * sizeof(double), solver_stream));
  checkCudaErrors(hipMallocAsync(&d_t, m * sizeof(double), solver_stream));

  // Initialize variables to evaluate convergence
  double x_error = 1e50;
  double x_error_init = 1e50;
  double* d_xprev = NULL;
  checkCudaErrors(hipMallocAsync(&d_xprev, m * sizeof(double), solver_stream));

  // Create descriptors for variables that will pass through cuSPARSE
  hipsparseSpMatDescr_t spDescrA;
  hipsparseDnVecDescr_t spDescrB;
  hipsparseDnVecDescr_t spDescrX0;
  hipsparseDnVecDescr_t spDescrP;
  hipsparseDnVecDescr_t spDescrNu;
  hipsparseDnVecDescr_t spDescrT;
  checkCudaErrors(hipsparseCreateCoo(&spDescrA, m, n, nnz, d_cooRowA, d_cooColA, d_cooValA_sorted, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
  checkCudaErrors(hipsparseCreateDnVec(&spDescrB, m, d_b, HIP_R_64F));
  checkCudaErrors(hipsparseCreateDnVec(&spDescrX0, m, d_x, HIP_R_64F));
  checkCudaErrors(hipsparseCreateDnVec(&spDescrP, m, d_p, HIP_R_64F));
  checkCudaErrors(hipsparseCreateDnVec(&spDescrNu, m, d_nu, HIP_R_64F));
  checkCudaErrors(hipsparseCreateDnVec(&spDescrT, m, d_t, HIP_R_64F));

  // Allocate work buffer for hipsparseSpMV
  size_t SpMVBuffSz;
  void* SpMVBuff;
  checkCudaErrors(hipsparseSpMV_bufferSize(
        cusparse_handle, 
        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        &eye, 
        spDescrA, 
        spDescrP, 
        &nil, 
        spDescrNu, 
        HIP_R_64F, 
        HIPSPARSE_MV_ALG_DEFAULT, 
        &SpMVBuffSz));
  checkCudaErrors(hipMallocAsync(&SpMVBuff, SpMVBuffSz * sizeof(char), solver_stream));

  // 1. r <- b - A*x_0.  Add bias with cuBLAS like in "NVIDIA_CUDA-11.4_Samples/7_CUDALibraries/conjugateGradient"
  checkCudaErrors(hipsparseSpMV( // A*x_0
        cusparse_handle, 
        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        &eye, 
        spDescrA, 
        spDescrX0, 
        &nil, 
        spDescrNu, // Use d_nu as temporary storage for result A*x_0 
        HIP_R_64F, 
        HIPSPARSE_MV_ALG_DEFAULT, 
        SpMVBuff)); 
  checkCudaErrors(hipblasDaxpy(cublas_handle, m, &nye, d_nu, 1, d_b, 1)); // r <- -A*x_0 + b
  
  // Calculate x_error_init for max_rel_error comparisons
  checkCudaErrors(hipblasDcopy(cublas_handle, m, d_x, 1, d_xprev, 1));
  checkCudaErrors(hipblasDaxpy(cublas_handle, m, &nye, d_b, 1, d_xprev, 1)); // initial solution guess stored in d_b
  checkCudaErrors(hipblasDnrm2(cublas_handle, m, d_xprev, 1, &x_error_init));

  std::cout << "FIRST NORM: " << x_error_init << std::endl;
  // 2. Set r_hat = r
  checkCudaErrors(hipblasDcopy(cublas_handle, m, d_b, 1, d_rhat, 1));

  // 3. Set initial values to scalars
  bool bConverged = false;
  int restarts = 0;
  double rho_curr = 1.;
  double rho_prev = 1.;
  double alpha = 1.;
  double omega = 1.;
  double beta = 0.;
  const double eps = 1e-21;

  // 4. Set initial values of vectors to zero
  checkCudaErrors(hipMemsetAsync(d_nu, 0, m * sizeof(double), solver_stream));
  checkCudaErrors(hipMemsetAsync(d_p, 0, m * sizeof(double), solver_stream));

  // 5. Start iterations
  const size_t max_iter = 1000;
  for(size_t k(0); k<max_iter; k++)
  {
    // 1. rho_i = (r_hat, r)
    checkCudaErrors(hipblasDdot(cublas_handle, m, d_rhat, 1, d_b, 1, &rho_curr));
    
    double norm_1 = 0.;
    double norm_2 = 0.;
    checkCudaErrors(hipblasDnrm2(cublas_handle, m, d_b, 1, &norm_1));
    checkCudaErrors(hipblasDnrm2(cublas_handle, m, d_rhat, 1, &norm_2));
    checkCudaErrors(hipStreamSynchronize(solver_stream)); // sync for 2. which happens on host
    // 2. beta = (rho_i / rho_{i-1}) * (alpha / omega_{i-1})
    beta = (rho_curr / (rho_prev+eps)) * (alpha / (omega+eps));

    // Numerical convergence trick
    const double cosTheta = rho_curr / norm_1 / norm_2;
    bool serious_breakdown = std::fabs(cosTheta) < 1e-8; 
    if(serious_breakdown && max_restarts > 0)
    {
      restarts++;
      if(restarts >= max_restarts){
        break;
      }
      std::cout << "[BiCGSTAB]: Restart at iteration: " << k << " norm: " << x_error <<" Initial norm: " << x_error_init << std::endl;
      checkCudaErrors(hipblasDcopy(cublas_handle, m, d_b, 1, d_rhat, 1));
      checkCudaErrors(hipblasDnrm2(cublas_handle, m, d_rhat, 1, &rho_curr));
      checkCudaErrors(hipStreamSynchronize(solver_stream)); 
      rho_curr *= rho_curr;
      rho_prev = 1.;
      alpha = 1.;
      omega = 1.;
      beta = (rho_curr / (rho_prev+eps)) * (alpha / (omega+eps));
    }

    // 3. p_i = r_{i-1} + beta(p_{i-1} - omega_{i-1}*nu_i)
    double nomega = -omega;
    checkCudaErrors(hipblasDaxpy(cublas_handle, m, &nomega, d_nu, 1, d_p, 1)); // p <- -omega_{i-1}*nu_i + p
    checkCudaErrors(hipblasDscal(cublas_handle, m, &beta, d_p, 1));            // p <- beta * p
    checkCudaErrors(hipblasDaxpy(cublas_handle, m, &eye, d_b, 1, d_p, 1));    // p <- r_{i-1} + p

    // 4. nu_i = A * p_i 
    checkCudaErrors(hipsparseSpMV(
          cusparse_handle,
          HIPSPARSE_OPERATION_NON_TRANSPOSE,
          &eye,
          spDescrA,
          spDescrP,
          &nil,
          spDescrNu,
          HIP_R_64F,
          HIPSPARSE_MV_ALG_DEFAULT,
          SpMVBuff));

    // 5. alpha = rho_i / (r_hat, nu_i)
    double alpha_den;
    checkCudaErrors(hipblasDdot(cublas_handle, m, d_rhat, 1, d_nu, 1, &alpha_den)); // alpha <- (r_hat, nu_i)
    checkCudaErrors(hipStreamSynchronize(solver_stream)); // sync for host division
    alpha = rho_curr / (alpha_den+eps); // alpha <- rho_i / alpha

    // 6. h = alpha*p_i + x_{i-1}
    checkCudaErrors(hipblasDcopy(cublas_handle, m, d_x, 1, d_xprev, 1)); // copy previous value for future norm calculation
    checkCudaErrors(hipblasDaxpy(cublas_handle, m, &alpha, d_p, 1, d_x, 1));

    // 7. If h accurate enough then quit
    checkCudaErrors(hipblasDaxpy(cublas_handle, m, &nye, d_x, 1, d_xprev, 1));
    checkCudaErrors(hipblasDnrm2(cublas_handle, m, d_xprev, 1, &x_error));
    checkCudaErrors(hipStreamSynchronize(solver_stream));

    if((x_error <= max_error) || (x_error / x_error_init <= max_rel_error))
    // if(x_error <= max_error)
    {
      std::cout << "  [BiCGSTAB]: Converged after " << k << " iterations" << std::endl;
      bConverged = true;
      break;
    }

    // 8. s = -alpha * nu_i + r_{i-1}
    const double nalpha = -alpha;
    checkCudaErrors(hipblasDaxpy(cublas_handle, m, &nalpha, d_nu, 1, d_b, 1));

    // 9. t = A * s
    checkCudaErrors(hipsparseSpMV(
          cusparse_handle,
          HIPSPARSE_OPERATION_NON_TRANSPOSE,
          &eye,
          spDescrA,
          spDescrB,
          &nil,
          spDescrT,
          HIP_R_64F,
          HIPSPARSE_MV_ALG_DEFAULT,
          SpMVBuff));
    
    // 10. omega_i = (t,s)/(t,t), variables alpha & beta no longer in use this iter
    double omega_num;
    double omega_den;
    checkCudaErrors(hipblasDdot(cublas_handle, m, d_t, 1, d_b, 1, &omega_num)); // alpha <- (t,s)
    checkCudaErrors(hipblasDnrm2(cublas_handle, m, d_t, 1, &omega_den));          // beta <- sqrt(t,t)
    checkCudaErrors(hipStreamSynchronize(solver_stream)); // sync for host arithmetic
    omega = omega_num / (omega_den * omega_den + eps);

    // 11. x_i = omega_i * s + h
    checkCudaErrors(hipblasDcopy(cublas_handle, m, d_x, 1, d_xprev, 1)); // copy previous value for future norm calculation
    checkCudaErrors(hipblasDaxpy(cublas_handle, m, &omega, d_b, 1, d_x, 1));

    // 12. If x_i accurate enough then quit
    checkCudaErrors(hipblasDaxpy(cublas_handle, m, &nye, d_x, 1, d_xprev, 1));
    checkCudaErrors(hipblasDnrm2(cublas_handle, m, d_xprev, 1, &x_error));
    checkCudaErrors(hipStreamSynchronize(solver_stream));

    if((x_error <= max_error) || (x_error / x_error_init <= max_rel_error))
    // if(x_error <= max_error)
    {
      std::cout << "[BiCGSTAB]: Converged after " << k << " iterations" << std::endl;;
      bConverged = true;
      break;
    }

    // 13. r_i = -omega_i * t + s
    nomega = -omega;
    checkCudaErrors(hipblasDaxpy(cublas_handle, m, &nomega, d_t, 1, d_b, 1));

    // Update *_prev values for next iteration
    rho_prev = rho_curr;
  }

  if( bConverged )
    std::cout <<  " Error norm (relative) = " << x_error << "/" << max_error << " (" << x_error/x_error_init  << "/" << max_rel_error << ")" << std::endl;
  else
    std::cout <<  "  [Poisson solver]: Iteration " << max_iter << ". Error norm (relative) = " << x_error << "/" << max_error << " (" << x_error/x_error_init  << "/" << max_rel_error << ")" << std::endl;


  // Copy result back to host
  checkCudaErrors(hipMemcpyAsync(h_x, d_x, m * sizeof(double), hipMemcpyDeviceToHost, solver_stream));

  // Cleanup memory alocated during BiCGSTAB
  checkCudaErrors(hipsparseDestroySpMat(spDescrA));
  checkCudaErrors(hipsparseDestroyDnVec(spDescrB));
  checkCudaErrors(hipsparseDestroyDnVec(spDescrX0));
  checkCudaErrors(hipsparseDestroyDnVec(spDescrP));
  checkCudaErrors(hipsparseDestroyDnVec(spDescrNu));
  checkCudaErrors(hipsparseDestroyDnVec(spDescrT));
  checkCudaErrors(hipFreeAsync(d_rhat, solver_stream));
  checkCudaErrors(hipFreeAsync(d_p, solver_stream));
  checkCudaErrors(hipFreeAsync(d_nu, solver_stream));
  checkCudaErrors(hipFreeAsync(d_t, solver_stream));
  checkCudaErrors(hipFreeAsync(d_xprev, solver_stream));
  checkCudaErrors(hipFreeAsync(SpMVBuff, solver_stream));


  // ------------------------------------------------------------------------------------------------------------------

  // Free device memory allocated for linear system
  checkCudaErrors(hipFreeAsync(d_cooValA, solver_stream));
  checkCudaErrors(hipFreeAsync(d_cooValA_sorted, solver_stream));
  checkCudaErrors(hipFreeAsync(d_cooRowA, solver_stream));
  checkCudaErrors(hipFreeAsync(d_cooColA, solver_stream));
  checkCudaErrors(hipFreeAsync(d_x, solver_stream));
  checkCudaErrors(hipFreeAsync(d_b, solver_stream));

  checkCudaErrors(hipStreamSynchronize(solver_stream));
  // Destroy CUDA stream and library handles
  checkCudaErrors(hipblasDestroy(cublas_handle)); 
  checkCudaErrors(hipsparseDestroy(cusparse_handle)); 
  checkCudaErrors(hipStreamDestroy(solver_stream));
}
